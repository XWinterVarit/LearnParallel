#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
void array_print(float* array, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", *(array+i));
    }
    printf("\n");
}
int main()
{

    char** bufferArray = (char**)malloc(sizeof(char*)*20);
    char A[] = "aaa";
    char B[] = "bbb";
    char C[] = "ccc";
    char D[] = "ddd";
    char E[] = "eee";
    *(bufferArray+0) = A;
    *(bufferArray+1) = B;
    *(bufferArray+2) = C;
    *(bufferArray+3) = D;
    *(bufferArray+4) = E;
    for (int i = 0; i < 4; ++i) {
        printf("buff array : %s\n", *(bufferArray+i));
    }

    char** d_bufferArray;
    hipMalloc(&d_bufferArray, )

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = (N + threadPerBlock - 1) / threadPerBlock;
    VecAdd<<<blockPerGrid, threadPerBlock>>>(d_A,d_B,d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



    array_print(h_A,3);
    array_print(h_B,3);
    array_print(h_C,3);
}