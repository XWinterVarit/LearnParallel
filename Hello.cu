
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
__global__ void hello (int receive) {
    int index = threadIdx.x;
    printf("Hello from thread : %d with receive value : %d\n", index, receive);
}
int main(int argc, char **argv) {
    printf("well\n");
    hello<<<1,2>>>(5);
    hipDeviceSynchronize();
    return 0;
}