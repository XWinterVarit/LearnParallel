
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BLOCK_DIM 32
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__global__ void matrixAdd (int *a, int *b, int *c);

// Random
void swap_int(int* a, int* b){ int tmp = *a; *a=*b; *b=tmp; }

int rand_int_hi_lo(int upper, int lower){
    return((rand() % (upper-lower+1)) + lower);
}

int rand_int(int a,int b)
{
    if (b > a) swap_int(&a,&b);
    return rand_int_hi_lo(a,b);
}
// End of Random

int* createMatrix (int row, int column) {
    return  (int*) malloc(sizeof(int) * row * column);
}
void generateValue_Matrix(int* Matrix, int rowsize, int columnsize, char options, int predata[20][20]){
    for (int inrow = 0; inrow < rowsize; inrow++) {
        for (int incolumn = 0; incolumn < columnsize; incolumn++) {
            switch (options) {
                case 's':
                    *(Matrix+inrow+(incolumn*rowsize)) = 0;
                    break;
                case 'p':
                    *(Matrix+inrow+(incolumn*rowsize)) = predata[inrow][incolumn];
                    break;
                case 'r':
                    *(Matrix+inrow+(incolumn*rowsize)) = rand_int(1,9);
                    break;
            }
        }
    }
}
void printMatrixM(int* Matrix, int rowsize, int columnsize) {
    for (int inrow = 0; inrow < rowsize; inrow++) {
        for (int incolumn = 0; incolumn < columnsize; incolumn++) {
            printf("%d ", *(Matrix+inrow+(incolumn*rowsize)));
        }
        printf("\n");
    }
}
__global__ void multiplication_Matrix (int *matrixA, int *matrixB, int *matrixC, int matrixA_rowsize, int matrixA_columnsize, int matrixB_rowsize, int matrixB_columnsize) {
    int matrixAnswer_rowsize = matrixA_rowsize;
    int matrixAnswer_columnsize = matrixB_columnsize;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < matrixAnswer_rowsize && col < matrixAnswer_columnsize) {
        int answer = 0;
        for (int incolumn_MatrixA = 0; incolumn_MatrixA < matrixA_columnsize; incolumn_MatrixA++) {
            answer += *(matrixA+row+(incolumn_MatrixA*matrixA_rowsize)) * *(matrixB+incolumn_MatrixA+(col*matrixB_rowsize));
        }
        *(matrixC+row+(col*matrixAnswer_rowsize)) = answer;
    }
}

int main() {
    time_t timestart = time(NULL);
    int matrixA_rowsize = 10000;
    int matrixA_columnsize = 10000;
    int matrixB_rowsize = 10000;
    int matrixB_columnsize = 10000;
    int matrixC_rowsize = matrixA_rowsize;
    int matrixC_columnsize = matrixB_columnsize;

    int* MatrixA = createMatrix(matrixA_rowsize,matrixA_columnsize);
    int* MatrixB = createMatrix(matrixB_rowsize,matrixB_columnsize);
    int* MatrixC = createMatrix(matrixC_rowsize,matrixC_columnsize);


    //int mA[20][20] = {{2,1,1,1,1},{1,1,1,1,1},{1,1,1,1,1},{1,1,1,1,1},{1,1,1,1,1}};
    //int mB[20][20] = {{1,1,1,1,1},{1,1,1,1,1},{1,1,1,1,1},{1,1,1,1,1},{1,1,1,1,5}};

    int mA[20][20] = {{3,4,2}};
    int mB[20][20] = {{13,9,7,15},{8,7,4,6},{6,4,0,3}};

    generateValue_Matrix(MatrixA,matrixA_rowsize,matrixA_columnsize,'r', mA);
    generateValue_Matrix(MatrixB,matrixB_rowsize,matrixB_columnsize,'r', mB);

    //printMatrixM(MatrixA,matrixA_rowsize, matrixA_columnsize);
    //printMatrixM(MatrixB,matrixB_rowsize, matrixB_columnsize);

    printf("Generate value completed!\n");
    int *dev_MatrixA, *dev_MatrixB, *dev_MatrixC;
    int size_MatrixA = matrixA_rowsize * matrixA_columnsize * sizeof(int);
    int size_MatrixB = matrixB_rowsize * matrixB_columnsize * sizeof(int);
    int size_MatrixC = matrixC_rowsize * matrixC_columnsize * sizeof(int);

    hipMalloc((void**)&dev_MatrixA, size_MatrixA);
    hipMalloc((void**)&dev_MatrixB, size_MatrixB);
    hipMalloc((void**)&dev_MatrixC, size_MatrixC);

    hipMemcpy (dev_MatrixA, MatrixA, size_MatrixA, hipMemcpyHostToDevice);
    hipMemcpy (dev_MatrixB, MatrixB, size_MatrixB, hipMemcpyHostToDevice);

    dim3 dimBlock (BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid ((int)ceil((matrixC_rowsize*1.0)/dimBlock.x),(int)ceil((matrixC_columnsize*1.0)/dimBlock.y));
    printf("thread per block is : %d\n", BLOCK_DIM*BLOCK_DIM);
    printf("block per grid is : %d , %d\n", (int)ceil((matrixC_rowsize*1.0)/dimBlock.x),(int)ceil((matrixC_columnsize*1.0)/dimBlock.y));

    multiplication_Matrix<<<dimGrid, dimBlock>>>(dev_MatrixA, dev_MatrixB, dev_MatrixC, matrixA_rowsize, matrixA_columnsize, matrixB_rowsize, matrixB_columnsize);
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //cudaMemcpy (MatrixA, dev_MatrixA, size, cudaMemcpyDeviceToHost);
    //cudaMemcpy (MatrixB, dev_MatrixB, size, cudaMemcpyDeviceToHost);
    //cudaMemcpy (MatrixC, dev_MatrixC, size_MatrixC, cudaMemcpyDeviceToHost);
    hipFree(dev_MatrixA); hipFree(dev_MatrixB); hipFree(dev_MatrixC);

    printf("---------------------------\n");

    //printMatrixM(MatrixA,matrixA_rowsize, matrixA_columnsize);
    //printMatrixM(MatrixB,matrixB_rowsize, matrixB_columnsize);

    //printMatrixM(MatrixC,matrixC_rowsize, matrixC_columnsize);
    printf("Calulate completed\n");
    printf("\nestimate using time : %.5f\n", (double)(time(NULL) - timestart));

}

__global__ void matrixAdd (int *a, int *b, int *c) {
    //int row = blockIdx.x * blockDim.x + threadIdx.x;
    //int col = blockIdx.y * blockDim.y + threadIdx.y;

    //int index = row + col * N;

    //if (row < N && col < N) {
        //*(a+index) = 5;
        //*(b+index) = 5;
    //    c[index] = a[index] + b[index];
    //}
}