
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BLOCK_DIM 1024





__global__ void matrixAdd (int *a, int N) {
    //int row = blockIdx.x * blockDim.x + threadIdx.x;
    //int col = blockIdx.y * blockDim.y + threadIdx.y;


    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        a[index] = index;
        //printf("Hello from blockidx %d  threadidx %d  index %d  a[index] %d \n", blockIdx.x, threadIdx.x, index, a[index]);

    }
    //if (row < N && col < N) {
    //*(a+index) = 5;
    //*(b+index) = 5;
    //    c[index] = a[index] + b[index];
    //}
}
int* createVector (int size, int inivalue) {
    int* vector = (int*) malloc(sizeof(int)*size);
    for (int i = 0; i < size; ++i) {
        vector[i] = inivalue;
    }
    return vector;
}
void readVector (int* vector, int size) {
    for (int i = 0; i < size; ++i) {
        printf("%d ",vector[i]);
    }
    printf("\n");
}
int main() {
    int dev = 0, driverVersion = 0, runtimeVersion = 0;

    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf("  Warp size:                                     %d\n", deviceProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
           deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);








    //int* test = (int*) malloc(sizeof(int)*4);
    int sizeVector = 1024;
    int memsizeVector = sizeof(int) * sizeVector;
    int* Vector = createVector(sizeVector,2);
    readVector(Vector,sizeVector);
/*
    *(test) = 0;
    *(test+1) = 0;    *(test+2) = 0;

    for (int i = 0; i < 3; ++i) {
        printf("%d ", test[i]);
    }
    printf("\n");
*/

    int *dev_Vector;


    hipMalloc((void**)&dev_Vector, memsizeVector);


    hipMemcpy (dev_Vector, Vector, memsizeVector, hipMemcpyHostToDevice);



    matrixAdd<<<1, BLOCK_DIM>>>(dev_Vector,sizeVector);
    hipDeviceSynchronize();

    hipMemcpy (Vector, dev_Vector, memsizeVector, hipMemcpyDeviceToHost);
    hipFree(dev_Vector);

    printf("---------------------------\n");


    readVector(Vector, sizeVector);

    printf("Calulate completed");
}

