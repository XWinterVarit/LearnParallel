#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define BLOCKDIM 1024
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
using namespace thrust;
using namespace thrust::placeholders;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

char FileName[] = "01.nt";
char OutputFileName[] = "b.txt";
//char DefineWord[] = "http://www.w3.org/2001/XMLSchema#string";
char DefineWord[] = "a";

long threadchucksize = 200000000; // aka file read buffer size 200 MB each
int WordCount;
__device__ size_t d_strlen (const char *str)
{
    return (*str) ? d_strlen(++str) + 1 : 0;
}
__device__ int d_strncmp(const char *ptr0, const char *ptr1, size_t len)
{
/*
    printf("print test.....  ");
    for (int i = 0; i < len; ++i) {
        printf("%c", *(ptr1+i));
    }
    printf("\n--");*/

    //printf("%s \n", ptr1);
/*

    int fast = len/sizeof(size_t) + 1;
    int offset = (fast-1)*sizeof(size_t);
    int current_block = 0;

    if( len <= sizeof(size_t)){ fast = 0; }


    size_t *lptr0 = (size_t*)ptr0;
    size_t *lptr1 = (size_t*)ptr1;

    while( current_block < fast ){
        if( (*(lptr0+current_block) ^ *(lptr1+current_block) )){
            int pos;

            for(pos = current_block*sizeof(size_t); pos < len ; ++pos ){
                if( (  *(ptr0+pos) ^ *(ptr1+pos)   ) || (  *(ptr0+pos) == 0) || (  *(ptr1+pos) == 0) ){
                    return  (int)((unsigned char) *(ptr0+pos) - (unsigned char) *(ptr1+pos));
                }
            }

        }

        ++current_block;
    }

    while( len > offset ){

        if( (  *(ptr0+offset) ^ *(ptr1+offset) )){
            return (int)((unsigned char) *(ptr0+offset) - (unsigned char) *(ptr1+offset));
        }
        ++offset;
    }
    return 0;
*/
/*
        for(; *ptr0 == *ptr1; ++ptr0, ++ptr1)
            if(*ptr0 == 0)
                return 0;
        return *(unsigned char *)ptr0 < *(unsigned char *)ptr1 ? -1 : 1;
*/
    while(len--)
        if(*ptr0++!=*ptr1++)
            return *(unsigned char*)(ptr0 - 1) - *(unsigned char*)(ptr1 - 1);
    return 0;


}
__device__ unsigned int string_search(long start, long end, char* target, char *buffer) {
    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        //printf("t dkmfdsfdfdspfdsfpodsfjkdpsof: %d \n", t);
        if (t == 0) {
            //if (i <= overflowRegion)
            found++;
        }
    }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__device__ unsigned int string_search_rr(long start, long end, char* target, char *buffer,int overflowStringSize, char options) {
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
/*
    int found = 0;
    for (long i = start; i < end +1 ; i++) {
        int charfail = 0;
        int wordindex = 0;
        //printf("show string : str1 : %c     word :  \n", *(buffer+i));
        for (int j = i; j < i + overflowStringSize && j < end + 1; ++j) {
            //printf("show string : str1 : %c     word : %c \n", *(buffer+j), *(target+wordindex));

            if (*(buffer+j) != *(target+wordindex))
                charfail = 1;

            wordindex++;
        }

        if (charfail == 0)
            found++;
        //printf("cal found : %d\n", found);
        //printf("----\n");

    }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
    */


    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end  ; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        //printf("t dkmfdsfdfdspfdsfpodsfjkdpsof: %d \n", t);
        if (t == 0 ) {
            //if (i <= overflowRegion)
            found++;
        }

    }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__global__ void cuda_stringsearch (long bufferstart, long bufferend, char* target, char* buffer, int* allcount, int overflowStringSize, long *answerVector) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("hello from thread %d\n", index);
    //printf("Hello from threads %d  Given word %s\n", index, target);

    long blocksize = 50000;
    long extendblocksize = blocksize + overflowStringSize - 2;
    long startpoint = index * blocksize;
    long endpoint = startpoint + blocksize  - 1;

    if (startpoint <= bufferend) {
        if (endpoint > bufferend)
            endpoint = bufferend;

        //int count = 10;
        int count = string_search_rr(startpoint, endpoint,target, buffer, overflowStringSize, 'd');
        //printf("threads %d count %d  getting data :  startpoint %ld  endpoint %ld  overflowStringSize %d\n", index,count,startpoint, endpoint, overflowStringSize);
        *(answerVector + index) = count;
        //*allcount += count;
    }

}

long* createVector (long size, long inivalue) {
    long* vector = (long*) malloc(sizeof(long)*size);
    for (long i = 0; i < size; ++i) {
        vector[i] = inivalue;
    }
    return vector;
}
void readVector (long* vector, long size) {
    for (long i = 0; i < size; ++i) {
        printf("%ld ",vector[i]);
    }
    printf("\n");
}
long sumVector (long* vector, long size) {
    long sum = 0;
    for (long i = 0; i < size; ++i) {
        sum+= vector[i];
    }
    return sum;
}
int main(int argc, char **argv) {


    FILE * pFile;
    long lSize;
    pFile = fopen ( FileName , "r" );
    if (pFile==NULL) {fputs ("File error",stderr); exit (1);}
    fseek (pFile , 0 , SEEK_END);
    lSize = ftell (pFile);
    rewind (pFile);
    long BufferSize = sizeof(char)*lSize;
    printf("Buffer index size %lu \n",BufferSize);

    int reverseoffset = 0;
    /* create threads */
    long endpoint = 0,startpoint = 0;
    int overflowStringSize = sizeof(DefineWord)/ sizeof(char) - 1;
    printf("Overflow String size : %d\n", overflowStringSize);



    FILE * outputFile;
    long lSize2;
    outputFile = fopen(OutputFileName, "a");
    if (outputFile==NULL) {fputs ("File error",stderr); exit (1);}


    int count = 0;
    int* countPTR = &count;
    int overflowRegion = threadchucksize - 1;
    while (1){
        //printf("precount  all count %d\n", count);

        char *buffer;
        startpoint = 0;
        endpoint = threadchucksize + overflowStringSize - 1;
        buffer = (char*) malloc (sizeof(char)*(threadchucksize + overflowStringSize));
        fseek (pFile , reverseoffset , SEEK_CUR);
        reverseoffset = -1 * (overflowStringSize - 1);

        fread (buffer,1,endpoint,pFile);
        if (BufferSize <= threadchucksize)
            endpoint = BufferSize;
        printf("This will send buffer start at %ld to %ld of all %ld\n", startpoint, endpoint, BufferSize);
        //int j = 0;
        //count += string_search(startpoint, endpoint, DefineWord, buffer);
        //printf("%s||| count : %d\n", buffer, count);
        //printf("-------\n");

        long size_answerVector = 4096;
        long* answerVector = createVector(size_answerVector,0);



        char *dev_buffer;
        int *dev_countPTR;
        char *dev_defineword;
        long *dev_answerVector;

        hipMalloc((void**)&dev_buffer, sizeof(char)*(threadchucksize + overflowStringSize));
        hipMalloc((void**)&dev_countPTR, sizeof(int));
        hipMalloc((void**)&dev_answerVector, sizeof(long)*size_answerVector);
        hipMalloc((void**)&dev_defineword, sizeof(DefineWord));

        hipMemcpy(dev_buffer, buffer, sizeof(char)*(threadchucksize + overflowStringSize), hipMemcpyHostToDevice);
        hipMemcpy(dev_defineword, DefineWord, sizeof(DefineWord), hipMemcpyHostToDevice);
        hipMemcpy(dev_countPTR, countPTR, sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(dev_answerVector, answerVector, sizeof(long)*size_answerVector, hipMemcpyHostToDevice);
        cuda_stringsearch<<<4,1024>>>(startpoint, endpoint, dev_defineword, dev_buffer, dev_countPTR, overflowStringSize, dev_answerVector);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //hipMemcpy (buffer, dev_buffer,sizeof(char)*(threadchucksize + overflowStringSize),hipMemcpyDeviceToHost);
        //hipMemcpy (countPTR, dev_countPTR, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy (answerVector, dev_answerVector, sizeof(long)*size_answerVector, hipMemcpyDeviceToHost);
        hipFree(dev_buffer); hipFree(dev_countPTR); hipFree(dev_defineword);
        hipFree(dev_answerVector);
/*
        for (int i = 0; i < 1000; ++i) {
            cuda_stringsearch(startpoint, endpoint, DefineWord, buffer, countPTR, overflowStringSize);
        }

        printf("---------Buffer after changed----------------------------------------------------------------\n");
        endpoint = threadchucksize - 1;
        if (endpoint > BufferSize)
            endpoint = BufferSize;
        printf("startpoint : %ld     endpoint : %ld   BufferSize : %ld \n", startpoint, endpoint, BufferSize);
        for (int i=startpoint;i <= endpoint; i++) {
            printf("%c", *(buffer + i));
            if (*(buffer + i) != '\0')
                fprintf(outputFile, "%c", *(buffer + i));
        }
        printf("\n");


        printf("-------------------------------------------------------------------------------------------\n");
*/
        //fprintf(outputFile, "%s",buffer);
        readVector(answerVector, size_answerVector);
        long iterationsum =  sumVector(answerVector, size_answerVector);

        count += iterationsum;
        printf("iteration count : %ld   all count %d\n", iterationsum, count);
        BufferSize = BufferSize - threadchucksize;

        free(buffer);
        if (BufferSize <= 0)
            break;
    }
    printf("all count : %d \n", count);
    fclose (pFile);
    fclose (outputFile);
    return EXIT_SUCCESS;
}