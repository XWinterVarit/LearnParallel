#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define BLOCKDIM 1024
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
using namespace thrust;
using namespace thrust::placeholders;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

const char FileName[] = "01.nt";
const char OutputFileName[] = "b.txt";
const char QuestionFileName[] = "q.txt";
//char DefineWord[] = "http://www.w3.org/2001/XMLSchema#string";
//char DefineWord[] = "a";

const long threadchucksize = 100000000; // threadchucksize / blocksize "must" <= size of answer vector ,aka file read buffer size 200 MB each
const long blocksize = 25000;

const int sizeofQuestionArray = 2048;
const int sizeofMaximumQuestionWord = 2048;

const long sizeofAnswerVector = 4096; // number of blocks * thread per block
const int NumberOfComputeBlock = 4;
const int NumberOfThreadsPerBlock = 1024;




int WordCount;
__device__ size_t d_strlen (const char *str)
{
    return (*str) ? d_strlen(++str) + 1 : 0;
}
__device__ int d_strncmp(const char *ptr0, const char *ptr1, size_t len)
{
/*
    printf("print test.....  ");
    for (int i = 0; i < len; ++i) {
        printf("%c", *(ptr1+i));
    }
    printf("\n--");*/

    //printf("%s \n", ptr1);
/*

    int fast = len/sizeof(size_t) + 1;
    int offset = (fast-1)*sizeof(size_t);
    int current_block = 0;

    if( len <= sizeof(size_t)){ fast = 0; }


    size_t *lptr0 = (size_t*)ptr0;
    size_t *lptr1 = (size_t*)ptr1;

    while( current_block < fast ){
        if( (*(lptr0+current_block) ^ *(lptr1+current_block) )){
            int pos;

            for(pos = current_block*sizeof(size_t); pos < len ; ++pos ){
                if( (  *(ptr0+pos) ^ *(ptr1+pos)   ) || (  *(ptr0+pos) == 0) || (  *(ptr1+pos) == 0) ){
                    return  (int)((unsigned char) *(ptr0+pos) - (unsigned char) *(ptr1+pos));
                }
            }

        }

        ++current_block;
    }

    while( len > offset ){

        if( (  *(ptr0+offset) ^ *(ptr1+offset) )){
            return (int)((unsigned char) *(ptr0+offset) - (unsigned char) *(ptr1+offset));
        }
        ++offset;
    }
    return 0;
*/
/*
        for(; *ptr0 == *ptr1; ++ptr0, ++ptr1)
            if(*ptr0 == 0)
                return 0;
        return *(unsigned char *)ptr0 < *(unsigned char *)ptr1 ? -1 : 1;
*/
    while(len--)
        if(*ptr0++!=*ptr1++)
            return *(unsigned char*)(ptr0 - 1) - *(unsigned char*)(ptr1 - 1);
    return 0;


}
__device__ unsigned int string_search(long start, long end, char* target, char *buffer) {
    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        //printf("t dkmfdsfdfdspfdsfpodsfjkdpsof: %d \n", t);
        if (t == 0) {
            //if (i <= overflowRegion)
            found++;
        }
    }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__device__ unsigned int string_search_rr(long start, long end, char* target, char *buffer,int overflowStringSize, char options) {

    unsigned int i;
    unsigned int found=0;

    for (i=start;i <= end  ; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        if (t == 0 ) {
            //if (i <= overflowRegion)
            found++;

            for (int j = i; j < i + d_strlen(target); ++j) {
                //printf("change at j : %d i : %d\n", j,i);
                *(buffer+j) = '$';
            }

        }

    }

    return found;
}
__global__ void cuda_stringsearch (long bufferstart, long bufferend, char* target, char* buffer, int* allcount, int overflowStringSize, long *answerVector) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("hello from thread %d\n", index);
    //printf("Hello from threads %d  Given word %s\n", index, target);

   // long blocksize = 500/*50000*/;
    long extendblocksize = blocksize + overflowStringSize - 2;
    long startpoint = index * blocksize;
    long endpoint = startpoint + blocksize  - 1;

    if (startpoint <= bufferend) {
        if (endpoint > bufferend)
            endpoint = bufferend;

        //int count = 10;
        int count = string_search_rr(startpoint, endpoint,target, buffer, overflowStringSize, 'd');
        //printf("threads %d count %d  getting data :  startpoint %ld  endpoint %ld  overflowStringSize %d\n", index,count,startpoint, endpoint, overflowStringSize);
        *(answerVector + index) = count;
        //*allcount += count;
    }

}

long* createVector (long size, long inivalue) {
    long* vector = (long*) malloc(sizeof(long)*size);
    for (long i = 0; i < size; ++i) {
        vector[i] = inivalue;
    }
    return vector;
}
void readVector (long* vector, long size) {
    for (long i = 0; i < size; ++i) {
        printf("%ld ",vector[i]);
    }
    printf("\n");
}
long sumVector (long* vector, long size) {
    long sum = 0;
    for (long i = 0; i < size; ++i) {
        sum+= vector[i];
    }
    return sum;
}
int main(int argc, char **argv) {


    FILE * pFile;
    long lSize;
    pFile = fopen ( FileName , "r" );
    if (pFile==NULL) {fputs ("File error",stderr); exit (1);}
    fseek (pFile , 0 , SEEK_END);
    lSize = ftell (pFile);
    rewind (pFile);
    long BufferSize = sizeof(char)*lSize;
    printf("Buffer index size %lu \n",BufferSize);

    int reverseoffset = 0;
    /* create threads */
    long endpoint = 0,startpoint = 0;




    FILE * questionFile;
    long lSizeQ;
    questionFile = fopen(QuestionFileName, "r");
    if (questionFile == NULL) {fputs ("File error", stderr); exit(1);}
    fseek(questionFile, 0, SEEK_END);
    lSizeQ = ftell(questionFile);
    rewind(questionFile);
    long QuestionBufferSize = sizeof(char)*lSizeQ;
    printf("Question Buffer index size %lu \n", QuestionBufferSize);
    char *Question_Buffer = (char*) malloc (lSizeQ);
    fread(Question_Buffer, 1, QuestionBufferSize, questionFile);
    printf("This is question file --------\n");

    //printf("%s\n", Question_Buffer);

    long start = 0, end = 0;
    int Question_maxLength = 0;
    char** questionArray = (char**) malloc(sizeof(char*)*sizeofQuestionArray);
    long* questionAnswer = (long*) malloc(sizeof(long)*sizeofQuestionArray);

    int questionCount = 0;
    for (int j = 0; j <= strlen(Question_Buffer); ++j) {
        end++;
        if (*(Question_Buffer+j) == '\n' || *(Question_Buffer+j) == '\0') {
            questionCount++;
            //piece = (char*) malloc(sizeof(char)*2048);
            //memcpy(piece, (Question_Buffer+start), end - start - 1);
            *(questionArray+questionCount) = (char*) malloc(sizeof(char)*sizeofAnswerVector);
            memcpy(*(questionArray+questionCount), (Question_Buffer+start), end - start - 1);
            *(questionAnswer+questionCount) = 0/*(long) strlen(*(questionArray+questionCount))*/;
            if (strlen(*(questionArray+questionCount)) > Question_maxLength)
                Question_maxLength = strlen(*(questionArray+questionCount));
            //printf("print piece %s|||\n", *(questionArray+questionCount));
            //printf("piece length : %lu \n", strlen(*(questionArray+questionCount)));
            start = end;

        }
    }

    for (int k = 1; k <= questionCount; ++k) {
        printf("element at : %d is : %s value is : %lu\n",  k, *(questionArray+k), *(questionAnswer+k));
    }

    printf("Question max length : %d\n", Question_maxLength);
    printf("Question elements count : %d\n", questionCount);
    free(Question_Buffer);
    printf("This is question file --------\n");

    int overflowStringSize = Question_maxLength /*- 1*/;
    printf("Overflow String size : %d\n", overflowStringSize);



    FILE * outputFile;
    long lSize2;
    outputFile = fopen(OutputFileName, "a");
    if (outputFile==NULL) {fputs ("File error",stderr); exit (1);}


    int count = 0;
    int* countPTR = &count;
    int overflowRegion = threadchucksize - 1;
    while (1){
        //printf("precount  all count %d\n", count);

        char *buffer;
        startpoint = 0;
        endpoint = threadchucksize + overflowStringSize - 1;
        buffer = (char*) malloc (sizeof(char)*(threadchucksize + overflowStringSize));
        fseek (pFile , reverseoffset , SEEK_CUR);
        reverseoffset = -1 * (overflowStringSize - 1);

        fread (buffer,1,endpoint,pFile);
        if (BufferSize <= threadchucksize)
            endpoint = BufferSize;
        printf("This will send buffer start at %ld to %ld of all %ld\n", startpoint, endpoint, BufferSize);

        //int j = 0;
        //count += string_search(startpoint, endpoint, DefineWord, buffer);
        //printf("%s||| count : %d\n", buffer, count);
        //printf("-------\n");




        char *dev_buffer;
        int *dev_countPTR;
        char *dev_defineword;
        long *dev_answerVector;

        hipMalloc((void**)&dev_buffer, sizeof(char)*(threadchucksize + overflowStringSize));
        hipMalloc((void**)&dev_countPTR, sizeof(int));

        hipMemcpy(dev_buffer, buffer, sizeof(char)*(threadchucksize + overflowStringSize), hipMemcpyHostToDevice);
        hipMemcpy(dev_countPTR, countPTR, sizeof(int),hipMemcpyHostToDevice);

        for (int question = 1; question <= questionCount; ++question) {
        //int question = 1;
            long size_answerVector = sizeofAnswerVector;
            long* answerVector = createVector(size_answerVector,0);
            hipMalloc((void**)&dev_answerVector, sizeof(long)*size_answerVector);
            hipMalloc((void**)&dev_defineword, sizeof(*(questionArray+question)));

            hipMemcpy(dev_answerVector, answerVector, sizeof(long)*size_answerVector, hipMemcpyHostToDevice);
            hipMemcpy(dev_defineword, *(questionArray+question), sizeof(*(questionArray+question)), hipMemcpyHostToDevice);
            //printf("iteration at question : %s\n", *(questionArray+question));

            cuda_stringsearch<<<NumberOfComputeBlock,NumberOfThreadsPerBlock>>>(startpoint, endpoint, dev_defineword, dev_buffer, dev_countPTR, overflowStringSize, dev_answerVector);
            hipDeviceSynchronize();
            hipMemcpy (answerVector, dev_answerVector, sizeof(long)*size_answerVector, hipMemcpyDeviceToHost);

            hipFree(dev_answerVector);
            hipFree(dev_defineword);

            //readVector(answerVector, size_answerVector);
            long iterationsum =  sumVector(answerVector, size_answerVector);

            *(questionAnswer+question) += iterationsum;

            //printf("iteration at question : %s   founded %ld\n", *(questionArray+question),  iterationsum);
            free(answerVector);
        }


        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        hipMemcpy (buffer, dev_buffer,sizeof(char)*(threadchucksize + overflowStringSize),hipMemcpyDeviceToHost);
        //hipMemcpy (countPTR, dev_countPTR, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(dev_buffer); hipFree(dev_countPTR);

        printf("---------saveing change buffered----------------------------------------------------------------\n");
        endpoint = threadchucksize - 1;
        if (endpoint > BufferSize)
            endpoint = BufferSize;
        //printf("startpoint : %ld     endpoint : %ld   BufferSize : %ld \n", startpoint, endpoint, BufferSize);
        for (int i=startpoint;i <= endpoint; i++) {
            if (*(buffer + i) != '\0' && *(buffer + i) != '$' )
                fprintf(outputFile, "%c", *(buffer + i));
            //printf("%c", *(buffer + i));
        }
        printf("\n");

        printf("-------------------------------------------------------------------------------------------\n");

        //fprintf(outputFile, "%s",buffer);

        BufferSize = BufferSize - threadchucksize;

        free(buffer);
        if (BufferSize <= 0)
            break;
    }
    for (int k = 1; k <= questionCount; ++k) {
        printf("element at : %d is : %s value is : %lu\n",  k, *(questionArray+k), *(questionAnswer+k));
    }
    fclose (pFile);
    fclose (outputFile);
    return EXIT_SUCCESS;
}