#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
using namespace thrust;
using namespace thrust::placeholders;


char FileName[] = "a.txt";
char OutputFileName[] = "b.txt";
char DefineWord[] = "Cat";
long threadchucksize = 49000000;
int WordCount;

__device__ unsigned int string_search(long start, long end, char* target, char *buffer) {
    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end; i++)
        if (strncmp(&buffer[i],target,strlen(target))==0) {
            //if (i <= overflowRegion)
            found++;
        }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__device__ unsigned int string_search_rr(long start, long end, char* target, char *buffer,int overflowStringSize, char options) {
    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end; i++)
        if (strncmp(&buffer[i],target,strlen(target))==0) {
            //if (i <= overflowRegion)
            found++;
            //*(buffer+i) = '$';

            for (int j = i; j < i + overflowStringSize ; j++) {
                *(buffer+j) = '$';
            }
        }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__global__ void cuda_stringsearch (long bufferstart, long bufferend, char* target, char* buffer, int* allcount, int overflowStringSize) {
    long blocksize = 49000;
    long extendblocksize = blocksize + overflowStringSize - 2;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    long startpoint = index * blocksize;
    long endpoint = startpoint + blocksize  - 1;

    if (startpoint <= bufferend) {
        if (endpoint > bufferend)
            endpoint = bufferend;
        int count = string_search_rr(startpoint, endpoint,target, buffer, overflowStringSize, 'd');
        *allcount += count;
        printf("*******************************************************************************************************************\n");
        printf("Hello from fake threads : %d   startpoint : %ld   logical_endpoint : %ld    sending_endpoint : %ld    bufferend : %ld count : %d \n", index, startpoint, startpoint + extendblocksize, endpoint, bufferend, count);

        for (int i = startpoint; i <= startpoint + extendblocksize ; ++i) {
            printf("%c", *(buffer+i));
        }

        printf("||\n");
    }

}

int main(int argc, char **argv) {

    FILE * pFile;
    long lSize;
    pFile = fopen ( FileName , "r" );
    if (pFile==NULL) {fputs ("File error",stderr); exit (1);}
    fseek (pFile , 0 , SEEK_END);
    lSize = ftell (pFile);
    rewind (pFile);
    long BufferSize = sizeof(char)*lSize;
    printf("Buffer index size %lu \n",BufferSize);

    int reverseoffset = 0;
    /* create threads */
    long endpoint = 0,startpoint = 0;
    int overflowStringSize = sizeof(DefineWord)/ sizeof(char) - 1;
    printf("Overflow String size : %d\n", overflowStringSize);



    FILE * outputFile;
    long lSize2;
    outputFile = fopen(OutputFileName, "a");
    if (outputFile==NULL) {fputs ("File error",stderr); exit (1);}


    int count = 0;
    int* countPTR = &count;
    int overflowRegion = threadchucksize - 1;
    while (1){
        char *buffer;
        startpoint = 0;
        endpoint = threadchucksize + overflowStringSize - 1;
        buffer = (char*) malloc (sizeof(char)*(threadchucksize + overflowStringSize));
        fseek (pFile , reverseoffset , SEEK_CUR);
        reverseoffset = -1 * (overflowStringSize - 1);

        fread (buffer,1,endpoint,pFile);
        if (BufferSize <= threadchucksize)
            endpoint = BufferSize;

        //int j = 0;
        //count += string_search(startpoint, endpoint, DefineWord, buffer);
        //printf("%s||| count : %d\n", buffer, count);
        //printf("-------\n");

        char *dev_buffer;
        int *dev_countPTR;

        hipMalloc((void**)&dev_buffer, sizeof(char)*(threadchucksize + overflowStringSize));
        hipMalloc((void**)&dev_countPTR, sizeof(int));

        hipMemcpy(dev_buffer, buffer, sizeof(char)*(threadchucksize + overflowStringSize), hipMemcpyHostToDevice);
        hipMemcpy(dev_countPTR, countPTR, sizeof(int),hipMemcpyHostToDevice);

        cuda_stringsearch<<<1,1024>>>(startpoint, endpoint, DefineWord, dev_buffer, dev_countPTR, overflowStringSize);
        hipMemcpy (buffer, dev_buffer,sizeof(char)*(threadchucksize + overflowStringSize),hipMemcpyDeviceToHost);
        hipMemcpy (countPTR, dev_countPTR, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(dev_buffer); hipFree(dev_countPTR);
/*
        for (int i = 0; i < 1000; ++i) {
            cuda_stringsearch(startpoint, endpoint, DefineWord, buffer, countPTR, overflowStringSize);
        }

        printf("---------Buffer after changed----------------------------------------------------------------\n");
        endpoint = threadchucksize - 1;
        if (endpoint > BufferSize)
            endpoint = BufferSize;
        printf("startpoint : %ld     endpoint : %ld   BufferSize : %ld \n", startpoint, endpoint, BufferSize);
        for (int i=startpoint;i <= endpoint; i++) {
            printf("%c", *(buffer + i));
            if (*(buffer + i) != '\0')
                fprintf(outputFile, "%c", *(buffer + i));
        }
        printf("\n");


        printf("-------------------------------------------------------------------------------------------\n");
*/
        //fprintf(outputFile, "%s",buffer);
        BufferSize = BufferSize - threadchucksize;

        free(buffer);
        if (BufferSize <= 0)
            break;
    }
    printf("all count : %d \n", *countPTR);
    fclose (pFile);
    fclose (outputFile);
    return EXIT_SUCCESS;
}