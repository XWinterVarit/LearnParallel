
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCKDIM 1024
/*
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
using namespace thrust;
using namespace thrust::placeholders;
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////Important Configuration/////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

const char FileName[] = "01.nt"; // the file must have
const char OutputFileName[] = "b.txt"; // the file must have, but must be empty
const char QuestionFileName[] = "line1"; // the file must have


// Threadchucksize mean "Size" in byte of part of the file that will be sent to the gpu memory
// not worry about setting size higher than real file, because the program will auto adjust it
// Example , if your gpu memory limit at 500 MB but your file is large such as 5 GB , you should set threadchucksize to 500 * 1,000,000byte = 500,000,000
// and your part of file will be sent 5000 MB/500 MB = 10 times
// if you sent and found some memory error, it can be the os or other software use vram too, so decrease threadchucksize until no error, such as from 500,000,000 change to 200,000,000
// suggestion, you should set size as beautiful ten-end number as 100000, 20000000, 50000000
const long threadchucksize = 300000000;


// this Blocksize not mean thread per block but mean "size" of data chuck that each thread will compute from the big global data chuck (that locate in gpu)
// warning that allThreadInUse multiply with blocksize must higher than threadchucksize, if not it will incorrect result
// Example : if threadchucksize = 100,000,000 (aka 100MB. chuck of file sent to gpu) and allThreadInUse = 4096 and blocksize = 25000
// then you must check that 25000 * 4096 > 100000000 ? which is 102,400,000 > 100,000,000 so it true and can be use
// suggestion, you should set size as threadchucksize % blocksize = 0, it will be bug free.
const long blocksize = 10000;


const int NumberOfComputeBlock = 30; // aka gridsize
const int NumberOfThreadsPerBlock = 1024; //(rely on your gpu spec)

// allThreadInUse is all concurrent thread that run in the gpu,
// it can be higher than physical cuda core on gpu, because gpu can queue it and make you feel like it concurrent
// but if allThreadInUse is much higher, the answer vector that collect answer from each thread will be larger. so threadoff
const int allThreadInUse = NumberOfThreadsPerBlock * NumberOfComputeBlock;
const long sizeofAnswerVector = allThreadInUse;


const int sizeofQuestionArray = 2048; // maximum list size of question, such as = 2048 mean this program support maximum 2048 question
const int sizeofMaximumQuestionWord = 2048; // maximum string size of each question, such as = 2048 mean each question can't be larger than 2048 byte


/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////



int WordCount;
__device__ size_t d_strlen (const char *str)
{
    return (*str) ? d_strlen(++str) + 1 : 0;
}
__device__ int d_strncmp(const char *ptr0, const char *ptr1, size_t len)
{
    while(len--)
        if(*ptr0++!=*ptr1++)
            return *(unsigned char*)(ptr0 - 1) - *(unsigned char*)(ptr1 - 1);
    return 0;
}
__device__ unsigned int string_search(long start, long end, char* target, char *buffer) {
    unsigned int i;
    unsigned int found=0;
    //printf("Receiveing : target : %s  buffer : %s\n", target, buffer);
    //printf("My Buffer: ");
    //for (i=start;i <= end; i++)
    //    printf("%c", *(buffer+i));
    //printf("\n");
    for (i=start;i <= end; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        //printf("t dkmfdsfdfdspfdsfpodsfjkdpsof: %d \n", t);
        if (t == 0) {
            //if (i <= overflowRegion)
            found++;
        }
    }
    //printf("overflowRegion : %d\n", overflowRegion);
    //printf("Receiveing Found : %d\n", found);
    return found;
}
__device__ unsigned int string_search_rr(long start, long end, char* target, char *buffer,int overflowStringSize, char options, char *changebuffer) {

    unsigned int i;
    unsigned int found=0;

    for (i=start;i <= end  ; i++) {
        int t = d_strncmp(&buffer[i], target, d_strlen(target));
        if (t == 0 ) {
            //if (i <= overflowRegion)
            found++;

            for (int j = i; j < i + d_strlen(target); ++j) {
                //printf("change at j : %d i : %d\n", j,i);
                *(changebuffer+j) = '$';
            }

        }

    }

    return found;
}
__global__ void cuda_stringsearch (long bufferstart, long bufferend, char* target, char* buffer, int* allcount, int overflowStringSize, long *answerVector, char* changebuffer) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("hello from thread %d\n", index);
/*  
  if (index == 1) {
        printf("GPU KERNEL :: Hello from threads %d  Given word %s  size %d \n", index, target, d_strlen(target));
    }
*/
    //printf("Hello from threads %d  Given word %s\n", index, target);

   // long blocksize = 500/*50000*/;
    long extendblocksize = blocksize + overflowStringSize - 2;
    long startpoint = index * blocksize;
    long endpoint = startpoint + blocksize  - 1;

    if (startpoint <= bufferend) {
        if (endpoint > bufferend)
            endpoint = bufferend;

        //int count = 10;
        int count = string_search_rr(startpoint, endpoint,target, buffer, overflowStringSize, 'd', changebuffer);
        //printf("threads %d count %d  getting data :  startpoint %ld  endpoint %ld  overflowStringSize %d\n", index,count,startpoint, endpoint, overflowStringSize);
        *(answerVector + index) = count;
        //*allcount += count;
    }

}

long* createVector (long size, long inivalue) {
    long* vector = (long*) malloc(sizeof(long)*size);
    for (long i = 0; i < size; ++i) {
        vector[i] = inivalue;
    }
    return vector;
}
void readVector (long* vector, long size) {
    for (long i = 0; i < size; ++i) {
        printf("%ld ",vector[i]);
    }
    printf("\n");
}
long sumVector (long* vector, long size) {
    long sum = 0;
    for (long i = 0; i < size; ++i) {
        sum+= vector[i];
    }
    return sum;
}
int main(int argc, char **argv) {

    time_t timestart = time(NULL);

    FILE * pFile;
    long lSize;
    pFile = fopen ( FileName , "r" );
    if (pFile==NULL) {fputs ("File error",stderr); exit (1);}
    fseek (pFile , 0 , SEEK_END);
    lSize = ftell (pFile);
    rewind (pFile);
    long BufferSize = sizeof(char)*lSize;
    printf("Buffer index size %lu \n",BufferSize);

    int reverseoffset = 0;
    /* create threads */
    long endpoint = 0,startpoint = 0;




    FILE * questionFile;
    long lSizeQ;
    questionFile = fopen(QuestionFileName, "r");
    if (questionFile == NULL) {fputs ("File error", stderr); exit(1);}
    fseek(questionFile, 0, SEEK_END);
    lSizeQ = ftell(questionFile);
    rewind(questionFile);
    long QuestionBufferSize = sizeof(char)*lSizeQ;
    printf("Question Buffer index size %lu \n", QuestionBufferSize);
    char *Question_Buffer = (char*) malloc (lSizeQ);
    fread(Question_Buffer, 1, QuestionBufferSize, questionFile);
    printf("This is question file --------\n");

    //printf("%s\n", Question_Buffer);

    long start = 0, end = 0;
    int Question_maxLength = 0;
    char** questionArray = (char**) malloc(sizeof(char*)*sizeofQuestionArray);
    long* questionAnswer = (long*) malloc(sizeof(long)*sizeofQuestionArray); // found word list

    int questionCount = 0;
    for (int j = 0; j <= strlen(Question_Buffer); ++j) {
        end++;
        if (*(Question_Buffer+j) == '\n' || *(Question_Buffer+j) == '\0') {
            questionCount++;
            //piece = (char*) malloc(sizeof(char)*2048);
            //memcpy(piece, (Question_Buffer+start), end - start - 1);
            *(questionArray+questionCount) = (char*) malloc(sizeof(char)*sizeofAnswerVector);
            memcpy(*(questionArray+questionCount), (Question_Buffer+start), end - start - 1);
            *(questionAnswer+questionCount) = 0; /* each question start founded = zero */               /*(long) strlen(*(questionArray+questionCount)) this commented code use to check if for loop work!*/
            if (strlen(*(questionArray+questionCount)) > Question_maxLength)
                Question_maxLength = strlen(*(questionArray+questionCount));
            //printf("print piece %s|||\n", *(questionArray+questionCount));
            //printf("piece length : %lu \n", strlen(*(questionArray+questionCount)));
            start = end;

        }
    }
/*
    for (int k = 1; k <= questionCount; ++k) {
        printf("element at : %d is : %s value is : %lu  length is %zu\n",  k, *(questionArray+k), *(questionAnswer+k), strlen(*(questionArray+k)));
    }
*/
    printf("Question max length : %d\n", Question_maxLength);
    printf("Question elements count : %d\n", questionCount);
    free(Question_Buffer);
    printf("This is question file --------\n");

    int overflowStringSize = Question_maxLength /*- 1*/;
    printf("Overflow String size : %d\n", overflowStringSize);



    FILE * outputFile;
    long lSize2;
    outputFile = fopen(OutputFileName, "a");
    if (outputFile==NULL) {fputs ("File error",stderr); exit (1);}


    int count = 0;
    int* countPTR = &count;
    int overflowRegion = threadchucksize - 1;
    while (1){
        //printf("precount  all count %d\n", count);

        char *buffer;
        startpoint = 0;
        endpoint = threadchucksize + overflowStringSize - 1;
        buffer = (char*) malloc (sizeof(char)*(threadchucksize + overflowStringSize));
        fseek (pFile , reverseoffset , SEEK_CUR);
        reverseoffset = -1 * (overflowStringSize - 1);

        fread (buffer,1,endpoint,pFile);
        if (BufferSize <= threadchucksize)
            endpoint = BufferSize;
        printf("This will send buffer start at %ld to %ld of all %ld\n", startpoint, endpoint, BufferSize);

        //int j = 0;
        //count += string_search(startpoint, endpoint, DefineWord, buffer);
        //printf("%s||| count : %d\n", buffer, count);
        //printf("-------\n");




        char *dev_buffer;
        char *dev_changebuffer;
        int *dev_countPTR;
        //char *dev_defineword;
        //long *dev_answerVector;

        hipMalloc((void**)&dev_buffer, sizeof(char)*(threadchucksize + overflowStringSize));
        hipMalloc((void**)&dev_changebuffer, sizeof(char)*(threadchucksize + overflowStringSize));
        hipMalloc((void**)&dev_countPTR, sizeof(int));

        hipMemcpy(dev_buffer, buffer, sizeof(char)*(threadchucksize + overflowStringSize), hipMemcpyHostToDevice);
        hipMemcpy(dev_changebuffer, buffer, sizeof(char)*(threadchucksize + overflowStringSize), hipMemcpyHostToDevice);

        hipMemcpy(dev_countPTR, countPTR, sizeof(int),hipMemcpyHostToDevice);

        for (int question = 1; question <= questionCount; ++question) {
        //int question = 1;
            long size_answerVector = sizeofAnswerVector;
            long* answerVector = createVector(size_answerVector,0);
	    char *dev_defineword;
	    long *dev_answerVector;

//	    printf("HOST :: starting iteration %d at question : %s  string length : %zu\n",question, *(questionArray+question),  strlen(*(questionArray+question)));
/*		for (int d = 0; d < strlen(*(questionArray+question)); d++) {
			printf("%c",*(*(questionArray+question)+d));
		}
		printf("\nend test \n");*/
            hipMalloc((void**)&dev_answerVector, sizeof(long)*size_answerVector);
            hipMalloc((void**)&dev_defineword, /*sizeof(char)**/ sizeofMaximumQuestionWord/*strlen(*(questionArray+question))*/);

            hipMemcpy(dev_answerVector, answerVector, sizeof(long)*size_answerVector, hipMemcpyHostToDevice);
            hipMemcpy(dev_defineword, *(questionArray+question), /*sizeof(char)**/ sizeofMaximumQuestionWord /*strlen(*(questionArray+question))*/, hipMemcpyHostToDevice);
            //printf("iteration at question : %s\n", *(questionArray+question));

            cuda_stringsearch<<<NumberOfComputeBlock,NumberOfThreadsPerBlock>>>(startpoint, endpoint, dev_defineword, dev_buffer, dev_countPTR, overflowStringSize, dev_answerVector, dev_changebuffer);
            hipDeviceSynchronize();
            hipMemcpy (answerVector, dev_answerVector, sizeof(long)*size_answerVector, hipMemcpyDeviceToHost);

            hipFree(dev_answerVector);
            hipFree(dev_defineword);
	    
            //readVector(answerVector, size_answerVector); //uncomment this to diagnostic answer vector matrix
            long iterationsum =  sumVector(answerVector, size_answerVector);

            *(questionAnswer+question) += iterationsum;

            printf("HOST :: Finish iteration %d at question : %s  temporary founded %ld\n\n",question, *(questionArray+question),  iterationsum);
            free(answerVector);
        }


        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        hipMemcpy (buffer, dev_changebuffer,sizeof(char)*(threadchucksize + overflowStringSize),hipMemcpyDeviceToHost);
        //cudaMemcpy (countPTR, dev_countPTR, sizeof(int), cudaMemcpyDeviceToHost);
        hipFree(dev_buffer); hipFree(dev_countPTR); hipFree(dev_changebuffer);

        printf("---------saving change buffered----------------------------------------------------------------\n");
        endpoint = threadchucksize - 1;
        if (endpoint > BufferSize)
            endpoint = BufferSize;
        //printf("startpoint : %ld     endpoint : %ld   BufferSize : %ld \n", startpoint, endpoint, BufferSize);
        for (int i=startpoint;i <= endpoint; i++) {
            if (*(buffer + i) != '\0' && *(buffer + i) != '$' )
                fprintf(outputFile, "%c", *(buffer + i));
            //printf("%c", *(buffer + i));
        }
        printf("\n");

        printf("-------------------------------------------------------------------------------------------\n");

        //fprintf(outputFile, "%s",buffer);

        BufferSize = BufferSize - threadchucksize;

        free(buffer);
        if (BufferSize <= 0)
            break;
    }
    for (int k = 1; k <= questionCount; ++k) {
        printf("element at : %d is : %s finally founded : %lu\n",  k, *(questionArray+k), *(questionAnswer+k));
    }
    fclose (pFile);
    fclose (outputFile);

    printf("\nestimate using time : %.2f\n", (double)(time(NULL) - timestart));

    return EXIT_SUCCESS;
}
