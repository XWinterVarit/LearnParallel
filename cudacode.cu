
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
void array_print(float* array, int size) {
    for (int i = 0; i < size; i++) {
        printf("%f ", *(array+i));
    }
    printf("\n");
}
int main()
{
    int N = 3;
    size_t size = N * sizeof(float);
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    *(h_A) = 2;
    *(h_A+1) = 4;
    *(h_A+2) = 6;

    *(h_B) = 2;
    *(h_B+1) = 4;
    *(h_B+2) = 6;

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadPerBlock = 1024;
    int blockPerGrid = (N + threadPerBlock - 1) / threadPerBlock;
    VecAdd<<<blockPerGrid, threadPerBlock>>>(d_A,d_B,d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



    array_print(h_A,3);
    array_print(h_B,3);
    array_print(h_C,3);
}